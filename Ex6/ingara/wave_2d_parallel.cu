#include "hip/hip_runtime.h"
// #define _XOPEN_SOURCE 600
// I get a compiler warning that the macro is already defined so i've commented
// it out

#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

// TASK: T1
// Include the cooperative groups library
// BEGIN: T1
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
namespace cg = cooperative_groups;
//  END: T1

// Convert 'struct timeval' into seconds in double prec. floating point
#define WALLTIME( t ) ( (double)( t ).tv_sec + 1e-6 * (double)( t ).tv_usec )

// Option to change numerical precision
typedef int64_t int_t;
typedef float   real_t; // I have been unable to pass make check if I use double,
                        // even though the implementation works with float.

// TASK: T1b
// Variables needed for implementation
// BEGIN: T1b

// Based on the output from the device info, max threads per block is 1024,
// which is 32x32 This could be determined dynamically in init_cuda, but I don't
// see the point since we know which GPUs we're using
#define BLOCKY 32
#define BLOCKX 32

// Simulation parameters: size, step count, and how often to save the state
int_t h_N = 128, h_M = 128, h_max_iteration = 1e6, h_snapshot_freq = 1e3;
#define SIM_DATA_SIZE ( ( h_M + 2 ) * ( h_N + 2 ) * sizeof ( real_t ) )
// I forgot to multiply by sizeof(real_t) one time, so I made this macro to
// avoid that mistake in the future

// Wave equation parameters, time step is derived from the space step
const real_t h_c = 1.0, h_dx = 1.0, h_dy = 1.0;
real_t       h_dt;

// Since the simulation and equation parameters are constant throughout the
// simulation, we can keep them in the device's constant memory for fast access
// by all threads. The h_ equivalents are copied to these in h_domain_initialize
__constant__ int_t  d_N, d_M;
__constant__ real_t d_c, d_dx, d_dy, d_dt;

// Buffers for three time steps, indexed with 2 ghost points for the boundary
// I've decided to make a wrapper struct to make it more clear which timestep is
// being used. The struct points the device memory allocated with hipMalloc,
// not host memory, but I've prepended it with h_ since it is used by the host
// and not the device
typedef struct Timesteps
{
    real_t *prv;
    real_t *cur;
    real_t *nxt;

} Timesteps;
Timesteps h_timesteps;

// Buffer for the host to copy device memory into when writing the results to
// file and an access macro for it
real_t *h_out;
#define h_U( i, j ) h_out[( ( i ) + 1 ) * ( h_N + 2 ) + ( j ) + 1]

// I have changed the macros to work with buffers being passed in to functions
// instead of being accessed globally. This is because I couldn't figure out how
// to have a globally accessible Timesteps struct on the device, so the host
// passes in the individual buffers to kernel calls as arguments, and the kernel
// will pass them along to any device functions that need them
#define U_prv( i, j ) prv[( ( i ) + 1 ) * ( d_N + 2 ) + ( j ) + 1]
#define U( i, j )     cur[( ( i ) + 1 ) * ( d_N + 2 ) + ( j ) + 1]
#define U_nxt( i, j ) nxt[( ( i ) + 1 ) * ( d_N + 2 ) + ( j ) + 1]

// Used to convert bytes to the more readable kibi-, mebi-, and gibibytes
#define BYTES_TO_KiB( bytes ) ( ( bytes ) / 1024.0 )
#define BYTES_TO_MiB( bytes ) ( ( bytes ) / ( 1024.0 * 1024.0 ) )
#define BYTES_TO_GiB( bytes ) ( ( bytes ) / ( 1024.0 * 1024.0 * 1024.0 ) )

// END: T1b

// I didn't end up using these
#define cudaErrorCheck( ans )                                                                      \
    {                                                                                              \
        gpuAssert ( ( ans ), __FILE__, __LINE__ );                                                 \
    }

inline void
gpuAssert ( hipError_t code, const char *file, int line, bool abort = true )
{
    if ( code != hipSuccess ) {
        fprintf ( stderr, "GPUassert: %s %s %d\n", hipGetErrorString ( code ), file, line );
        if ( abort ) {
            exit ( code );
        }
    }
}

// Save the present time step in a numbered file under 'data/'
void
h_domain_save ( int_t step )
{
    char filename[256];
    sprintf ( filename, "data/%.5ld.dat", step );
    FILE *out = fopen ( filename, "wb" );
    for ( int_t i = 0; i < h_M; i++ ) {
        fwrite ( &h_U ( i, 0 ), sizeof ( real_t ), h_N, out );
    }
    fclose ( out );
}

// TASK: T4
// Get rid of all the memory allocations
void
h_domain_finalize ( void )
{
    // BEGIN: T4
    free ( h_out );
    hipFree ( h_timesteps.prv );
    hipFree ( h_timesteps.cur );
    hipFree ( h_timesteps.nxt );
    // END: T4
}

// Rotate the time step buffers.
void
h_move_buffer_window ( void )
{
    real_t *temp    = h_timesteps.prv;
    h_timesteps.prv = h_timesteps.cur;
    h_timesteps.cur = h_timesteps.nxt;
    h_timesteps.nxt = temp;
}

// TASK: T6
// Neumann (reflective) boundary condition
// BEGIN: T6
__device__ void
d_boundary_condition ( int t_i, int t_j, real_t *cur )
{
    if ( t_j == 0 ) {
        U ( t_i, -1 ) = U ( t_i, 1 );
    }

    if ( t_j == ( d_N - 1 ) ) {
        U ( t_i, d_N ) = U ( t_i, d_N - 2 );
    }

    if ( t_i == 0 ) {
        U ( -1, t_j ) = U ( 1, t_j );
    }

    if ( t_i == ( d_M - 1 ) ) {
        U ( d_M, t_j ) = U ( d_M - 2, t_j );
    }
}
// END: T6

// TASK: T5
// Integration formula

__global__ void
d_time_step ( real_t *prv, real_t *cur, real_t *nxt )
{
    // BEGIN; T5
    cg::thread_block tb = cg::this_thread_block ();

    dim3 b_idx = tb.group_index ();
    dim3 b_dim = tb.group_dim ();
    dim3 t_idx = tb.thread_index ();

    int t_i = b_idx.x * b_dim.x + t_idx.x;
    int t_j = b_idx.y * b_dim.y + t_idx.y;

    d_boundary_condition ( t_i, t_j, cur );

    // Make sure all boundary conditions have been applied before calculating the
    // timestep
    tb.sync ();

    U_nxt ( t_i, t_j ) = -U_prv ( t_i, t_j ) + 2.0 * U ( t_i, t_j )
                       + ( d_dt * d_dt * d_c * d_c ) / ( d_dx * d_dy )
                             * ( U ( t_i - 1, t_j ) + U ( t_i + 1, t_j ) + U ( t_i, t_j - 1 )
                                 + U ( t_i, t_j + 1 ) - 4.0 * U ( t_i, t_j ) );
    // END: T5
}

// TASK: T7
// Main time integration.
void
h_simulate ( void )
{
    // BEGIN: T7
    // Go through each time step
    int_t grid_x = h_N / BLOCKX;
    int_t grid_y = h_M / BLOCKY;
    dim3  block ( BLOCKX, BLOCKY );
    dim3  grid ( grid_x, grid_y );

    // Originally, I wanted to perform all of the simulation steps in a loop on
    // the GPU, i.e. boundary condition, time step, and move buffer window, only
    // exiting device execution after snapshot_freq iterations to write the data
    // to file, but I couldn't get it to work, unfortunately.
    for ( int_t iteration = 0; iteration <= h_max_iteration; iteration += 1 ) {
        if ( ( iteration % h_snapshot_freq ) == 0 ) {
            hipMemcpy ( h_out, h_timesteps.cur, SIM_DATA_SIZE, hipMemcpyDeviceToHost );
            h_domain_save ( iteration / h_snapshot_freq );
        }

        // Derive step t+1 from steps t and t-1
        d_time_step<<<grid, block>>> ( h_timesteps.prv, h_timesteps.cur, h_timesteps.nxt );

        // Rotate the time step buffers
        h_move_buffer_window ();
    }
    // END: T7
}

// TASK: T8
// GPU occupancy
void
h_occupancy ( void )
{
    // BEGIN: T8
    int block_size;
    int min_grid_size;
    int max_active_blocks;

    hipOccupancyMaxPotentialBlockSize ( &min_grid_size, &block_size, d_time_step, 0, 0 );
    hipOccupancyMaxActiveBlocksPerMultiprocessor ( &max_active_blocks, d_time_step, block_size,
                                                    0 );

    int            device;
    hipDeviceProp_t props;
    hipGetDevice ( &device );
    hipGetDeviceProperties ( &props, device );

    int   grid_size = ( h_M + 2 ) * ( h_N + 2 );
    float occupancy = ( (float)max_active_blocks * block_size / props.warpSize )
                    / ( (float)props.maxThreadsPerMultiProcessor / props.warpSize );

    printf ( "\nGrid size set to %d\nMin grid size is %d\nBlock size is %d\nMax "
             "active blocks is "
             "%d\nDevice's max thread count per SM is %d\nDevice's warp size is "
             "%d\nTheoretical "
             "occupancy: %f\n",
             grid_size, min_grid_size, block_size, max_active_blocks,
             props.maxThreadsPerMultiProcessor, props.warpSize, occupancy );
    // END: T8
}

// TASK: T2
// Make sure at least one CUDA-capable device exists
static bool
h_init_cuda ( void )
{
    // BEGIN: T2
    int dev_count;
    hipGetDeviceCount ( &dev_count );
    printf ( "Device count: %d\n", dev_count );

    if ( dev_count <= 0 ) {
        return false;
    } else {
        hipSetDevice ( 0 );
    }

    hipDeviceProp_t dev_props;
    for ( int i = 0; i < dev_count; ++i ) {
        hipError_t ret = hipGetDeviceProperties ( &dev_props, i );
        if ( ret == hipErrorInvalidDevice ) {
            return false;
        }

        printf ( "\nName: %s\n", dev_props.name );
        printf ( "Compute capability: %d.%d\n", dev_props.major, dev_props.minor );
        printf ( "Multiprocessors: %d\n", dev_props.multiProcessorCount );
        printf ( "Warp size: %d\n", dev_props.warpSize );
        printf ( "Global memory: %.3fGiB\n", BYTES_TO_GiB ( dev_props.totalGlobalMem ) );
        printf ( "Per-block shared memory: %.3fKiB\n",
                 BYTES_TO_KiB ( dev_props.sharedMemPerBlock ) );
        printf ( "Per-block registers: %d\n", dev_props.regsPerBlock );
        printf ( "\nMax threads per block: %d\n", dev_props.maxThreadsPerBlock );
        printf ( "Max threads dim: %d x %d x %d\n", dev_props.maxThreadsDim[0],
                 dev_props.maxThreadsDim[1], dev_props.maxThreadsDim[2] );
        printf ( "Max gid size: %d x %d x %d\n", dev_props.maxGridSize[0], dev_props.maxGridSize[1],
                 dev_props.maxGridSize[2] );
    }

    return true;
    // END: T2
}

// TASK: T3
// Set up our three buffers, and fill two with an initial perturbation
__global__ void
d_init_timesteps ( real_t *prv, real_t *cur )
{
    // Calculating the initial values on the device, instead of on the host and
    // then copying it to device memory, should be faster (though it might not
    // necessarily be, but the necessary profiling to find out is outside the
    // scope of the exercise, in my opinion)
    cg::thread_block tb = cg::this_thread_block ();

    dim3 b_idx = tb.group_index ();
    dim3 b_dim = tb.group_dim ();
    dim3 t_idx = tb.thread_index ();

    int t_i = b_idx.x * b_dim.x + t_idx.x;
    int t_j = b_idx.y * b_dim.y + t_idx.y;

    real_t delta = sqrt ( ( ( t_i - d_M / 2.0 ) * ( t_i - d_M / 2.0 ) ) / (real_t)d_M
                          + ( ( t_j - d_N / 2.0 ) * ( t_j - d_N / 2.0 ) ) / (real_t)d_N );

    real_t val         = exp ( -4.0 * delta * delta );
    U_prv ( t_i, t_j ) = U ( t_i, t_j ) = val;
}

void
h_domain_initialize ( void )
{
    // BEGIN: T3

    bool locate_cuda = h_init_cuda ();
    if ( !locate_cuda ) {
        printf ( "Failed to init CUDA\n" );
        exit ( EXIT_FAILURE );
    }

    // Buffer on the host for writing the results to file
    h_out = (real_t *)malloc ( SIM_DATA_SIZE );

    hipMalloc ( (void **)&h_timesteps.prv, SIM_DATA_SIZE );
    hipMalloc ( (void **)&h_timesteps.cur, SIM_DATA_SIZE );
    hipMalloc ( (void **)&h_timesteps.nxt, SIM_DATA_SIZE );

    // Set the time step for 2D case
    h_dt = h_dx * h_dy / ( h_c * sqrt ( h_dx * h_dx + h_dy * h_dy ) );

    // Copy all relevant values for the simulation and equation to the device
    hipMemcpyToSymbol(HIP_SYMBOL( d_N), &h_N, sizeof ( int_t ) );
    hipMemcpyToSymbol(HIP_SYMBOL( d_M), &h_M, sizeof ( int_t ) );
    hipMemcpyToSymbol(HIP_SYMBOL( d_c), &h_c, sizeof ( real_t ) );
    hipMemcpyToSymbol(HIP_SYMBOL( d_dx), &h_dx, sizeof ( real_t ) );
    hipMemcpyToSymbol(HIP_SYMBOL( d_dy), &h_dy, sizeof ( real_t ) );
    hipMemcpyToSymbol(HIP_SYMBOL( d_dt), &h_dt, sizeof ( real_t ) );

    int_t grid_x = h_N / BLOCKX;
    int_t grid_y = h_M / BLOCKY;

    dim3 grid ( grid_x, grid_y );
    dim3 block ( BLOCKX, BLOCKY );
    d_init_timesteps<<<grid, block>>> ( h_timesteps.prv, h_timesteps.cur );

    // END: T3
}

int
main ( void )
{
    // Set up the initial state of the domain
    h_domain_initialize ();

    struct timeval t_start, t_end;
    gettimeofday ( &t_start, NULL );

    h_simulate ();

    gettimeofday ( &t_end, NULL );

    printf ( "\nTotal elapsed time: %lf seconds\n", WALLTIME ( t_end ) - WALLTIME ( t_start ) );

    h_occupancy ();

    // Clean up and shut down
    h_domain_finalize ();
    exit ( EXIT_SUCCESS );
}
