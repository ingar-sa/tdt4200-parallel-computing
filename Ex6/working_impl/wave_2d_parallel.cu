#include "hip/hip_runtime.h"
// #define _XOPEN_SOURCE 600
// I get a compiler warning that macro is already defined in the cuda headers, so i've commented it
// out
#include <hip/hip_runtime_api.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

#if DO_DEBUG == 1
#define PrintfDbg(...) printf(__VA_ARGS__)
#else
#define PrintfDbg(...)
#endif

// TASK: T1
// Include the cooperative groups library
// BEGIN: T1
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
namespace cg = cooperative_groups;
//  END: T1

// Convert 'struct timeval' into seconds in double prec. floating point
#define WALLTIME(t) ((double)(t).tv_sec + 1e-6 * (double)(t).tv_usec)

// Option to change numerical precision
typedef int64_t int_t;
typedef float   real_t;

// TASK: T1b
// Variables needed for implementation
// BEGIN: T1b

// Based on the output from the device info, max threads per block is 1024, which is 32x32
// This could be determined dynamically in init_cuda, but I don't see the point since we know which
// GPUs we're using
#define BLOCKY 32
#define BLOCKX 32

// Simulation parameters: size, step count, and how often to save the state
int_t h_N = 128, h_M = 128, h_max_iteration = 10000, h_snapshot_freq = 200;
#define SIM_DATA_SIZE ((h_M + 2) * (h_N + 2) * sizeof(real_t))
// I forgot to multiply by sizeof(real_t) one time, so I made this macro to avoid that mistake in
// the future

// Wave equation parameters, time step is derived from the space step
const real_t h_c = 1.0, h_dx = 1.0, h_dy = 1.0;
real_t       h_dt;

// Since the simulation and equation parameters are constant throughout the simulation, we can keep
// them in constant memory for fast access by all threads
__constant__ int_t  d_N, d_M, d_max_iteration, d_snapshot_freq;
__constant__ real_t d_c, d_dx, d_dy, d_dt;

// Buffers for three time steps, indexed with 2 ghost points for the boundary
// real_t *buffers[3] = { NULL, NULL, NULL };
typedef struct Timesteps
{
    real_t *prv;
    real_t *cur;
    real_t *nxt;

} Timesteps;
// I've decided to make a wrapper struct to make it more clear which timestep is
// being accessed. There is one on the host for allocating, freeing, and copying the
// computational results from the GPU into host memory (h_out). The struct is copied to the GPU in
// domain_initialize
Timesteps            h_timesteps;
__device__ Timesteps d_timesteps;
// TODO(ingar): Determine if this is an appropriate way to do this, or if the buffers should just be
// passed in at kernel launch

// Buffer for the host to use when writing the results to file and access macro for it
real_t *h_out;
#define h_U(i, j) h_out[((i) + 1) * (h_N + 2) + (j) + 1]

#define U_prv(i, j) timesteps.prv[((i) + 1) * (d_N + 2) + (j) + 1]
#define U(i, j)     timesteps.cur[((i) + 1) * (d_N + 2) + (j) + 1]
#define U_nxt(i, j) timesteps.nxt[((i) + 1) * (d_N + 2) + (j) + 1]

// #define U_p(i, j) d_timesteps.prv[((i) + 1) * (d_N + 2) + (j) + 1]
// #define U_c(i, j) d_timesteps.cur[((i) + 1) * (d_N + 2) + (j) + 1]
// #define U_n(i, j) d_timesteps.nxt[((i) + 1) * (d_N + 2) + (j) + 1]

#define U_p(i, j) prv[((i) + 1) * (d_N + 2) + (j) + 1]
#define U_c(i, j) cur[((i) + 1) * (d_N + 2) + (j) + 1]
#define U_n(i, j) nxt[((i) + 1) * (d_N + 2) + (j) + 1]

hipDeviceProp_t gpu;
#define BYTES_TO_KiB(bytes) ((bytes) / 1024.0)
#define BYTES_TO_MiB(bytes) ((bytes) / (1024.0 * 1024.0))
#define BYTES_TO_GiB(bytes) ((bytes) / (1024.0 * 1024.0 * 1024.0))

// END: T1b

#define cudaErrorCheck(ans)                                                                        \
    {                                                                                              \
        gpuAssert((ans), __FILE__, __LINE__);                                                      \
    }

inline void
gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if(code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if(abort) {
            exit(code);
        }
    }
}

// Save the present time step in a numbered file under 'data/'
void
domain_save(int_t step)
{
    char filename[256];
    sprintf(filename, "data/%.5ld.dat", step);
    FILE *out = fopen(filename, "wb");
    for(int_t i = 0; i < h_M; i++) {
        for(int_t j = 0; j < h_N; ++j) {
            if(h_U(i, j) > 0.01) {
                // PrintfDbg("i, j, val: %ld, %ld, %f\n", i, j, h_U(i, j));
            }
        }
        fwrite(&h_U(i, 0), sizeof(real_t), h_N, out);
    }
    fclose(out);
}

// TASK: T4
// Get rid of all the memory allocations
void
domain_finalize(void)
{
    // BEGIN: T4
    free(h_out);
    hipFree(h_timesteps.prv);
    hipFree(h_timesteps.cur);
    hipFree(h_timesteps.nxt);
    // END: T4
}

// Rotate the time step buffers.
#if 0
__device__ void
d_move_buffer_window()
{
    real_t *temp    = d_timesteps.prv;
    d_timesteps.prv = d_timesteps.cur;
    d_timesteps.cur = d_timesteps.nxt;
    d_timesteps.nxt = temp;
}
#endif

void
h_move_buffer_window(void)
{
    real_t *temp    = h_timesteps.prv;
    h_timesteps.prv = h_timesteps.cur;
    h_timesteps.cur = h_timesteps.nxt;
    h_timesteps.nxt = temp;
}

// TASK: T6
// Neumann (reflective) boundary condition
// BEGIN: T6
__device__ void
d_boundary_condition(int t_i, int t_j, real_t *cur)
{
    if(t_j == 0) {
        U_c(t_i, -1) = U_c(t_i, 1);
        // PrintfDbg("Thread %d, %d performing boundary condition\n", t_i, t_j);
    }

    if(t_j == (d_N - 1)) {
        // PrintfDbg("Thread %d, %d performing boundary condition\n", t_i, t_j);
        U_c(t_i, d_N) = U_c(t_i, d_N - 2);
    }

    if(t_i == 0) {
        // PrintfDbg("Thread %d, %d performing boundary condition\n", t_i, t_j);
        U_c(-1, t_j) = U_c(1, t_j);
    }

    if(t_i == (d_M - 1)) {
        // PrintfDbg("Thread %d, %d performing boundary condition\n", t_i, t_j);
        U_c(d_M, t_j) = U_c(d_M - 2, t_j);
    }
}
// END: T6

// TASK: T5
// Integration formula
// BEGIN; T5

#if 0
__global__ void
d_time_step(void)
{
    cg::thread_block tb = cg::this_thread_block();

    dim3 b_idx = tb.group_index();
    dim3 b_dim = tb.group_dim();
    dim3 t_idx = tb.thread_index();

    int t_i = b_idx.x * b_dim.x + t_idx.x;
    int t_j = b_idx.y * b_dim.y + t_idx.y;

    if(tb.thread_rank() == 0) {
        // printf("Performing time steps\n");
    }
    for(int_t iteration = 0; iteration < d_snapshot_freq; ++iteration) {
        tb.sync();
        d_boundary_condition(t_i, t_j);

        tb.sync();
        U_n(t_i, t_j) = -U_p(t_i, t_j) + 2.0 * U_c(t_i, t_j)
                      + (d_dt * d_dt * d_c * d_c) / (d_dx * d_dy)
                            * (U_c(t_i - 1, t_j) + U_c(t_i + 1, t_j) + U_c(t_i, t_j - 1)
                               + U_c(t_i, t_j + 1) - 4.0 * U_c(t_i, t_j));

        tb.sync();
        if(tb.thread_rank() == 0) {
            d_move_buffer_window();
        }
    }
}
#endif

#if 0
__global__ void
d_time_step(void)
{
    cg::thread_block tb = cg::this_thread_block();

    dim3 b_idx = tb.group_index();
    dim3 b_dim = tb.group_dim();
    dim3 t_idx = tb.thread_index();

    int t_i = b_idx.x * b_dim.x + t_idx.x;
    int t_j = b_idx.y * b_dim.y + t_idx.y;

    tb.sync();
    d_boundary_condition(t_i, t_j);

    tb.sync();
    U_n(t_i, t_j) = -U_p(t_i, t_j) + 2.0 * U_c(t_i, t_j)
                  + (d_dt * d_dt * d_c * d_c) / (d_dx * d_dy)
                        * (U_c(t_i - 1, t_j) + U_c(t_i + 1, t_j) + U_c(t_i, t_j - 1)
                           + U_c(t_i, t_j + 1) - 4.0 * U_c(t_i, t_j));

    tb.sync();
    if(tb.thread_rank() == 0) {
        d_move_buffer_window();
    }
}
#endif


#if 1
__global__ void
d_time_step(real_t *prv, real_t *cur, real_t *nxt)
{
    cg::thread_block tb = cg::this_thread_block();

    dim3 b_idx = tb.group_index();
    dim3 b_dim = tb.group_dim();
    dim3 t_idx = tb.thread_index();

    int t_i = b_idx.x * b_dim.x + t_idx.x;
    int t_j = b_idx.y * b_dim.y + t_idx.y;

    tb.sync();
    d_boundary_condition(t_i, t_j, cur);

    tb.sync();
    U_n(t_i, t_j) = -U_p(t_i, t_j) + 2.0 * U_c(t_i, t_j)
                  + (d_dt * d_dt * d_c * d_c) / (d_dx * d_dy)
                        * (U_c(t_i - 1, t_j) + U_c(t_i + 1, t_j) + U_c(t_i, t_j - 1)
                           + U_c(t_i, t_j + 1) - 4.0 * U_c(t_i, t_j));
    tb.sync();
}
#endif

// END: T5

// TASK: T7
// Main time integration.
// TODO(ingar): Make everything run on gpu then copy data to host asynchronously and call save
// function?
void
simulate(void)
{
    // BEGIN: T7
    // Go through each time step
    int_t grid_x = h_N / BLOCKX;
    int_t grid_y = h_M / BLOCKY;
    dim3  block(BLOCKX, BLOCKY);
    dim3  grid(grid_x, grid_y);

    for(int_t iteration = 0; iteration <= h_max_iteration; iteration += 1) {
        if((iteration % h_snapshot_freq) == 0) {
            // PrintfDbg("Saving domain %ld\n", iteration);
            //  hipMemcpyToSymbol(HIP_SYMBOL(&h_timesteps), &d_timesteps, sizeof(Timesteps),
            //  hipMemcpyDeviceToHost);

            hipMemcpy(h_out, h_timesteps.cur, SIM_DATA_SIZE, hipMemcpyDeviceToHost);
            domain_save(iteration / h_snapshot_freq);
        }

        // Derive step t+1 from steps t and t-1
        // boundary_condition();
        // printf("Performing time step\n");
        d_time_step<<<grid, block>>>(h_timesteps.prv, h_timesteps.cur, h_timesteps.nxt);

        // Rotate the time step buffers
        h_move_buffer_window();
    }
}
// END: T7


// TASK: T3
// Set up our three buffers, and fill two with an initial perturbation

__global__ void
d_init_timesteps(real_t *prv, real_t *cur)
{
    cg::thread_block tb = cg::this_thread_block();

    dim3 b_idx = tb.group_index();
    dim3 b_dim = tb.group_dim();
    dim3 t_idx = tb.thread_index();

    int t_i = b_idx.x * b_dim.x + t_idx.x;
    int t_j = b_idx.y * b_dim.y + t_idx.y;

    real_t delta = sqrt(((t_i - d_M / 2.0) * (t_i - d_M / 2.0)) / (real_t)d_M
                        + ((t_j - d_N / 2.0) * (t_j - d_N / 2.0)) / (real_t)d_N);

    real_t val    = exp(-4.0 * delta * delta);
    U_p(t_i, t_j) = U_c(t_i, t_j) = val;

    if(val > 0.1) {
        // PrintfDbg("U(%d, %d): %f\n", t_i, t_j, U(t_i, t_j));
    }
}

// TASK: T8
// GPU occupancy
void
occupancy(void)
{
    // BEGIN: T8
    ;
    // END: T8
}

// TASK: T2
// Make sure at least one CUDA-capable device exists
static bool
init_cuda(void)
{
    // BEGIN: T2
    int dev_count;
    hipGetDeviceCount(&dev_count);
    printf("Device count: %d\n", dev_count);

    if(dev_count <= 0) {
        return false;
    } else {
        hipSetDevice(0);
    }

    for(int i = 0; i < dev_count; ++i) {
        hipError_t ret = hipGetDeviceProperties(&gpu, i);
        if(ret == hipErrorInvalidDevice) {
            return false;
        }

        printf("Name: %s\n", gpu.name);
        printf("Compute capability: %d.%d\n", gpu.major, gpu.minor);
        printf("Multiprocessors: %d\n", gpu.multiProcessorCount);
        printf("Warp size: %d\n", gpu.warpSize);
        printf("Global memory: %.3fGiB\n", BYTES_TO_GiB(gpu.totalGlobalMem));
        printf("Per-block shared memory: %.3fKiB\n", BYTES_TO_KiB(gpu.sharedMemPerBlock));
        printf("Per-block registers: %d\n", gpu.regsPerBlock);
        printf("\nMax threads per block: %d\n", gpu.maxThreadsPerBlock);
        printf("Max threads dim: %d x %d x %d\n", gpu.maxThreadsDim[0], gpu.maxThreadsDim[1],
               gpu.maxThreadsDim[2]);
        printf("Max gid size: %d x %d x %d\n", gpu.maxGridSize[0], gpu.maxGridSize[1],
               gpu.maxGridSize[2]);
    }

    return true;
    // END: T2
}
void
domain_initialize(void)
{
    // BEGIN: T3
    bool locate_cuda = init_cuda();
    if(!locate_cuda) {
        printf("Failed to init CUDA\n");
        exit(EXIT_FAILURE);
    }

    // Buffer on the host for writing the results to file
    h_out = (real_t *)malloc(SIM_DATA_SIZE);

    // Allocate the memory so that it is referenced by the host-side Timesteps struct, and then
    // copy the result to the GPU
    hipMalloc((void **)&h_timesteps.prv, SIM_DATA_SIZE);
    hipMalloc((void **)&h_timesteps.cur, SIM_DATA_SIZE);
    hipMalloc((void **)&h_timesteps.nxt, SIM_DATA_SIZE);
    hipMemcpyToSymbol(HIP_SYMBOL(d_timesteps), &h_timesteps, sizeof(Timesteps));
    // hipMemcpyToSymbol defaults to hipMemcpyHostToDevice

    // Set the time step for 2D case
    h_dt = h_dx * h_dy / (h_c * sqrt(h_dx * h_dx + h_dy * h_dy));

    // Copy all of the variables for the simulation and equation to the GPU
    hipMemcpyToSymbol(HIP_SYMBOL(d_N), &h_N, sizeof(int_t));
    hipMemcpyToSymbol(HIP_SYMBOL(d_M), &h_M, sizeof(int_t));
    hipMemcpyToSymbol(HIP_SYMBOL(d_max_iteration), &h_max_iteration, sizeof(int_t));
    hipMemcpyToSymbol(HIP_SYMBOL(d_snapshot_freq), &h_snapshot_freq, sizeof(int_t));

    hipMemcpyToSymbol(HIP_SYMBOL(d_c), &h_c, sizeof(real_t));
    hipMemcpyToSymbol(HIP_SYMBOL(d_dx), &h_dx, sizeof(real_t));
    hipMemcpyToSymbol(HIP_SYMBOL(d_dy), &h_dy, sizeof(real_t));
    hipMemcpyToSymbol(HIP_SYMBOL(d_dt), &h_dt, sizeof(real_t));

    int_t grid_x = h_N / BLOCKX;
    int_t grid_y = h_M / BLOCKY;
    // PrintfDbg("grid_x (%ld), grid_y (%ld)\n", grid_x, grid_y);

    dim3 block(BLOCKX, BLOCKY);
    dim3 grid(grid_x, grid_y);
    d_init_timesteps<<<grid, block>>>(h_timesteps.prv, h_timesteps.cur);
    //  d_init_timesteps<<<grid, block>>>();

    // END: T3
}

int
main(void)
{
    // Set up the initial state of the domain
    init_cuda();
    domain_initialize();
    // domain_save(0);
    PrintfDbg("Domain succesfully initialized. Running simulation\n");

    struct timeval t_start, t_end;
    gettimeofday(&t_start, NULL);
    simulate();
    gettimeofday(&t_end, NULL);

    printf("Total elapsed time: %lf seconds\n", WALLTIME(t_end) - WALLTIME(t_start));

#if 0
    occupancy();
#endif

    // Clean up and shut down
    domain_finalize();
    PrintfDbg("Domain succesfully finalized. The simulation has been completed\n");
    exit(EXIT_SUCCESS);
}
