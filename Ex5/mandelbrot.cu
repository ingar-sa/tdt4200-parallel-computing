#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_vector_types.h>

/* Problem size */
#define XSIZE 2560
#define YSIZE 2048

/* Divide the problem into blocks of BLOCKX x BLOCKY threads */
#define BLOCKY 32
#define BLOCKX 32
#define MAXITER                                                                                    \
    255 // you may want to increase this
        //
double xleft  = -2.01;
double xright = 1;
double yupper, ylower;
double ycenter = 1e-6;
double step;

int host_pixel[XSIZE * YSIZE];
int device_pixel[XSIZE * YSIZE];

typedef struct
{
    float real, imag;
} complex_t;

#define PIXEL(i, j) ((i) + (j) * XSIZE)

typedef unsigned char uchar;
// save 24-bits bmp file, buffer must be in bmp format: upside-down
void
savebmp(const char *name, uchar *buffer, int x, int y)
{
    FILE *f = fopen(name, "wb");
    if(!f) {
        printf("Error writing image to disk.\n");
        return;
    }
    unsigned int size       = x * y * 3 + 54;
    uchar        header[54] = { 'B',
                                'M',
                                (uchar)(size & 255),
                                (uchar)((size >> 8) & 255),
                                (uchar)((size >> 16) & 255),
                                (uchar)(size >> 24),
                                0,
                                0,
                                0,
                                0,
                                54,
                                0,
                                0,
                                0,
                                40,
                                0,
                                0,
                                0,
                                (uchar)(x & 255),
                                (uchar)(x >> 8),
                                0,
                                0,
                                (uchar)(y & 255),
                                (uchar)(y >> 8),
                                0,
                                0,
                                1,
                                0,
                                24,
                                0,
                                0,
                                0,
                                0,
                                0,
                                0,
                                0,
                                0,
                                0,
                                0,
                                0,
                                0,
                                0,
                                0,
                                0,
                                0,
                                0,
                                0,
                                0,
                                0,
                                0,
                                0,
                                0,
                                0,
                                0 };
    fwrite(header, 1, 54, f);
    fwrite(buffer, 1, x * y * 3, f);
    fclose(f);
}

void
host_calculate()
{
    for(int j = 0; j < YSIZE; j++) {
        for(int i = 0; i < XSIZE; i++) {
            /* Calculate the number of iterations until divergence for
            each pixel.
            If divergence never happens, return MAXITER */
            complex_t c, z, temp;
            int       iter = 0;
            c.real         = (xleft + step * i);
            c.imag         = (yupper - step * j);
            z              = c;
            while(z.real * z.real + z.imag * z.imag < 4.0) {
                temp.real = z.real * z.real - z.imag * z.imag + c.real;
                temp.imag = 2.0 * z.real * z.imag + c.imag;
                z         = temp;
                if(++iter == MAXITER) {
                    break;
                }
            }
            host_pixel[PIXEL(i, j)] = iter;
        }
    }
}

// given iteration number, set a color
void
fancycolour(uchar *p, int iter)
{
    if(iter == MAXITER)
        ;
    else if(iter < 8) {
        p[0] = 128 + iter * 16;
        p[1] = p[2] = 0;
    } else if(iter < 24) {
        p[0] = 255;
        p[1] = p[2] = (iter - 8) * 16;
    } else if(iter < 160) {
        p[0] = p[1] = 255 - (iter - 24) * 2;
        p[2]        = 255;
    } else {
        p[0] = p[1] = (iter - 160) * 2;
        p[2]        = 255 - (iter - 160) * 2;
    }
}

// Get system time to microsecond precision
// ostensibly, similar to MPI_Wtime),
// returns time in seconds
double
walltime(void)
{
    static struct timeval t;
    gettimeofday(&t, NULL);
    return (t.tv_sec + 1e-6 * t.tv_usec);
}

// ********** SUBTASK1: Create kernel device_calculate ******************/
// Insert code here
// Hint: Use _global_ for the kernal function to be executed on the GPU.
// Also set up a single grid with a 2D thread block
// ********** SUBTASK1 END ***********************************************/

struct mb_args
{
    double xleft;
    double step;
    double yupper;
};

__global__ void
mandelbrot_gpu(int *pixels, struct mb_args *args)
{
    int t_i = blockIdx.x * blockDim.x + threadIdx.x;
    int t_j = blockIdx.y * blockDim.y + threadIdx.y;

    complex_t c, z, temp;
    int       iter = 0;
    c.real         = (args->xleft + args->step * t_i);
    c.imag         = (args->yupper - args->step * t_j);
    z              = c;

    while(z.real * z.real + z.imag * z.imag < 4.0) {
        temp.real = z.real * z.real - z.imag * z.imag + c.real;
        temp.imag = 2.0 * z.real * z.imag + c.imag;
        z         = temp;
        if(++iter == MAXITER) {
            break;
        }
    }

    pixels[PIXEL(t_i, t_j)] = iter;
}

int
main(int argc, char **argv)
{
    if(argc == 1) {
        puts("Usage: MANDEL n");
        puts("n decides whether image should be written to disk (1=yes, 0 = no) ");
        return 0;
    }

    double start;
    double hosttime   = 0;
    double devicetime = 0;
    double memtime    = 0;

    hipDeviceProp_t p;
    hipSetDevice(0);

    hipGetDeviceProperties(&p, 0);
    printf("Using GPU: %s\n", p.name);
    printf("Device compute capability: %d.%d\n", p.major, p.minor);

    /* Calculate the range in the y-axis such that we preserve the
    aspect ratio */
    step   = (xright - xleft) / XSIZE;
    yupper = ycenter + (step * YSIZE) / 2;
    ylower = ycenter - (step * YSIZE) / 2;

    /* Host calculates image */
    start = walltime();
    host_calculate();
    hosttime += walltime() - start;

    //********** SUBTASK2: Set up device memory ***************************/
    // Insert code here
    /********** SUBTASK2 END **********************************************/
    start = walltime();

    dim3 block(BLOCKX, BLOCKY);
    dim3 grid(XSIZE / BLOCKX, YSIZE / BLOCKY);

    int            *gpu_pixels;
    struct mb_args *gpu_args;

    hipMalloc((void **)&gpu_pixels, XSIZE * YSIZE * sizeof(*gpu_pixels));
    hipMalloc((void **)&gpu_args, sizeof(struct mb_args));

    struct mb_args args = { xleft, step, yupper };
    hipMemcpy(gpu_args, &args, sizeof(struct mb_args), hipMemcpyHostToDevice);

    //********* SUBTASK3: Execute the kernel on the device ************/
    // Insert code here
    //********** SUBTASK3 END *****************************************/
    devicetime += walltime() - start;
    start = walltime();

    mandelbrot_gpu<<<grid, block>>>(gpu_pixels, gpu_args);

    //***** SUBTASK4: Transfer the result from device to device_pixel[][]*/
    // Insert code here
    //********** SUBTASK4 END ******************************************/
    memtime += walltime() - start;
    hipMemcpy(device_pixel, gpu_pixels, XSIZE * YSIZE * sizeof(*gpu_pixels),
               hipMemcpyDeviceToHost);

    /****** SUBTASK5: Free the device memory also ************************/
    // Insert code here
    /********** SUBTASK5 END ******************************************/
    hipFree(gpu_pixels);
    hipFree(gpu_args);

    int errors = 0;
    /* check if result is correct */
    for(int i = 0; i < XSIZE; i++) {
        for(int j = 0; j < YSIZE; j++) {
            int diff = host_pixel[PIXEL(i, j)] - device_pixel[PIXEL(i, j)];
            if(diff < 0) {
                diff = -diff;
            }
            /* allow +-1 difference */
            if(diff > 1) {
                if(errors < 10) {
                    printf("Error on pixel %d %d: expected %d, found %d\n ", i, j,
                           host_pixel[PIXEL(i, j)], device_pixel[PIXEL(i, j)]);
                } else if(errors == 10) {
                    puts("...");
                }
                errors++;
            }
        }
    }

    if(errors > 0) {
        printf("Found %d errors.\n", errors);
    } else {
        puts("Device calculations are correct.");
    }

    printf("\n");
    printf("Host time: %7.3f ms\n", hosttime * 1e3);
    printf("Device calculation: %7.3f ms\n", devicetime * 1e3);
    printf("Copy result: %7.3f ms\n", memtime * 1e3);

    if(strtol(argv[1], NULL, 10) != 0) {
        /* create nice image from iteration counts. take care to create it
        upside
        down (bmp format) */
        unsigned char *buffer = (unsigned char *)calloc(XSIZE * YSIZE * 3, 1);
        for(int i = 0; i < XSIZE; i++) {
            for(int j = 0; j < YSIZE; j++) {
                int p = ((YSIZE - j - 1) * XSIZE + i) * 3;
                fancycolour(buffer + p, device_pixel[PIXEL(i, j)]);
            }
        }
        /* write image to disk */
        savebmp("mandel1.bmp", buffer, XSIZE, YSIZE);
    }
    return 0;
}
